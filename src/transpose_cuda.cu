#include <sputnik/sputnik.h>
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <c10/cuda/CUDAStream.h>
#include <hipsparse.h>
#include "error_check.h"

torch::Tensor allocate_transpose_workspace(hipsparseHandle_t* handle,
        int m, int n, int nonzeros, 
        torch::Tensor values, 
        torch::Tensor row_offsets,
        torch::Tensor column_indices, 
        torch::Tensor output_values, 
        torch::Tensor output_row_offsets,
        torch::Tensor output_column_indices
    ) {

    // Calculate the buffer size.
    size_t buffer_size = 0;
    CUSPARSE_CALL(hipsparseCsr2cscEx2_bufferSize(
        *handle, m, n, nonzeros, 
        values.data_ptr<float>(), 
        row_offsets.data_ptr<int>(),
        column_indices.data_ptr<int>(), 
        output_values.data_ptr<float>(), 
        output_row_offsets.data_ptr<int>(), 
        output_column_indices.data_ptr<int>(),
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
        HIPSPARSE_CSR2CSC_ALG1, &buffer_size));

    // Allocate the temporary buffer. Round up to the nearest float for the size of the buffer.
    int buffer_size_signed = (buffer_size + sizeof(float) - 1) / sizeof(float);
    
    auto options = torch::TensorOptions()
                        .dtype(torch::kFloat32)
                        .device(torch::kCUDA, 0);

    torch::Tensor workspace = torch::zeros({buffer_size_signed}, options);

    return workspace;
}

void csr_transpose(int m, int n,
                   torch::Tensor values, 
                   torch::Tensor row_offsets,
                   torch::Tensor column_indices,
                   torch::Tensor output_values,
                   torch::Tensor output_row_offsets,
                   torch::Tensor output_column_indices) {

    hipsparseHandle_t handle = NULL;
    CUSPARSE_CALL(hipsparseCreate(&handle));

    int nonzeros = values.size(0);

    auto options = torch::TensorOptions()
                                        .dtype(torch::kFloat32)
                                        .layout(torch::kStrided)
                                        .device(torch::kCUDA, values.device().index())
                                        .requires_grad(true);
    
    torch::Tensor workspace = allocate_transpose_workspace(&handle, m, n, nonzeros, 
                                                        values, 
                                                        row_offsets, 
                                                        column_indices, 
                                                        output_values, 
                                                        output_row_offsets, 
                                                        output_column_indices);

    // Launch the kernel.
    CUSPARSE_CALL(hipsparseCsr2cscEx2(
        handle, m, n, nonzeros, 
        values.data_ptr<float>(), 
        row_offsets.data_ptr<int>(),
        column_indices.data_ptr<int>(), 
        output_values.data_ptr<float>(), 
        output_row_offsets.data_ptr<int>(), 
        output_column_indices.data_ptr<int>(),
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
        HIPSPARSE_CSR2CSC_ALG1, workspace.data_ptr<float>()));

    hipsparseDestroy(handle);
}