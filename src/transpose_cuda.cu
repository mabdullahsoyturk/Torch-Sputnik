#include <sputnik/sputnik.h>
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <c10/cuda/CUDAStream.h>
#include <hipsparse.h>
#include "error_check.h"

torch::Tensor allocate_transpose_workspace(hipsparseHandle_t* handle,
        int m, int n, int nonzeros, 
        torch::Tensor values, 
        torch::Tensor row_offsets,
        torch::Tensor column_indices, 
        torch::Tensor output_values, 
        torch::Tensor output_row_offsets,
        torch::Tensor output_column_indices
    ) {

    // Calculate the buffer size.
    size_t buffer_size = 0;
    CUSPARSE_CALL(hipsparseCsr2cscEx2_bufferSize(
        *handle, m, n, nonzeros, 
        values.data_ptr<float>(), 
        row_offsets.data_ptr<int>(),
        column_indices.data_ptr<int>(), 
        output_values.data_ptr<float>(), 
        output_row_offsets.data_ptr<int>(), 
        output_column_indices.data_ptr<int>(),
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
        HIPSPARSE_CSR2CSC_ALG1, &buffer_size));

    // Allocate the temporary buffer. Round up to the nearest float for the size of the buffer.
    int buffer_size_signed = (buffer_size + sizeof(float) - 1) / sizeof(float);
    
    auto options = torch::TensorOptions()
                        .dtype(torch::kFloat32)
                        .device(torch::kCUDA, 0);

    torch::Tensor workspace = torch::zeros(buffer_size_signed, options);

    return workspace;
}

void csr_transpose(int m, int n, torch::Tensor nnzs,
                   torch::Tensor values, 
                   torch::Tensor row_offsets,
                   torch::Tensor column_indices,
                   torch::Tensor output_values,
                   torch::Tensor output_row_offsets,
                   torch::Tensor output_column_indices) {

    hipsparseHandle_t handle = NULL;
    CUSPARSE_CALL(hipsparseCreate(&handle));

    int dim_offset = nnzs.size(0) - 1;
    int replication = dim_offset == 1 ? nnzs.size(0) : 1;

    int* nonzeros = nnzs.data_ptr<int>();
    int sum = 0;

    for (int idx = 0; idx < replication; ++idx) {
        torch::Tensor workspace = allocate_transpose_workspace(&handle, m, n, nonzeros[idx], 
                                                          values + sum, 
                                                          row_offsets + (m + 1) * idx, 
                                                          column_indices + sum, 
                                                          output_values + sum, 
                                                          output_row_offsets + (m + 1) * idx, 
                                                          output_column_indices + sum);

        // Launch the kernel.
        CUSPARSE_CALL(hipsparseCsr2cscEx2(
            handle, m, n, nonzeros[idx], 
            values.data_ptr<float>() + sum, 
            row_offsets.data_ptr<int>() + (m + 1) * idx,
            column_indices.data_ptr<int>() + sum, 
            output_values.data_ptr<float>() + sum, 
            output_row_offsets.data_ptr<int>() + (m + 1) * idx, 
            output_column_indices.data_ptr<int>() + sum,
            HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
            HIPSPARSE_CSR2CSC_ALG1, workspace.data_ptr<float>()));

        sum += nonzeros[idx];
    }

    hipsparseDestroy(handle);
    hipDeviceSynchronize();
}