#include <sputnik/sputnik.h>
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAStream.h>
#include <hipsparse.h>
#include <vector>
#include "error_check.h"

torch::Tensor allocate_transpose_workspace_many(hipsparseHandle_t* handle,
        int m, int n, int nonzeros, 
        torch::Tensor values, 
        torch::Tensor row_offsets,
        torch::Tensor column_indices, 
        torch::Tensor output_values, 
        torch::Tensor output_row_offsets,
        torch::Tensor output_column_indices
    ) {

    // Calculate the buffer size.
    size_t buffer_size = 0;
    CUSPARSE_CALL(hipsparseCsr2cscEx2_bufferSize(
        *handle, m, n, nonzeros, 
        values.data_ptr<float>(), 
        row_offsets.data_ptr<int>(),
        column_indices.data_ptr<int>(), 
        output_values.data_ptr<float>(), 
        output_row_offsets.data_ptr<int>(), 
        output_column_indices.data_ptr<int>(),
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
        HIPSPARSE_CSR2CSC_ALG1, &buffer_size));

    // Allocate the temporary buffer. Round up to the nearest float for the size of the buffer.
    int buffer_size_signed = (buffer_size + sizeof(float) - 1) / sizeof(float);
    
    auto options = torch::TensorOptions()
                        .dtype(torch::kFloat32)
                        .device(torch::kCUDA, values.device().index());

    torch::Tensor workspace = torch::zeros({buffer_size_signed}, options);

    return workspace;
}

std::vector<torch::Tensor> csr_transpose_many_mask(int b, int m, int n,
                    torch::Tensor nonzeros,
                    torch::Tensor values, 
                    torch::Tensor row_offsets,
                    torch::Tensor column_indices) {
    /*--- CHECKS ---*/
    assert(row_offsets.dim() == 1); // Row offsets should have 1 dimension
    assert(column_indices.dim() == 1); // Column indices should have 1 dimension
    //assert(row_offsets.size(0) == m + 1); // Expected m+1 row offsets

    hipsparseHandle_t handle = at::cuda::getCurrentCUDASparseHandle();

    int replication = values.size(0);

    int num_heads = replication / b;
    int column_indices_out_size = 0;
    int max_nonzeros = -1;

    for(int i = 0; i < nonzeros.size(0); i++) {
        int nonzero = nonzeros[i].item<int>();
        if(nonzero > max_nonzeros) {
            max_nonzeros = nonzero;
        }

        column_indices_out_size += nonzero;
    }

    auto values_options = torch::TensorOptions()
                                        .dtype(torch::kFloat32)
                                        .layout(torch::kStrided)
                                        .device(torch::kCUDA, values.device().index());

    auto index_options = torch::TensorOptions()
                                        .dtype(torch::kInt32)
                                        .layout(torch::kStrided)
                                        .device(torch::kCUDA, values.device().index());
    

    torch::Tensor output_values = torch::zeros({replication, max_nonzeros}, values_options);
    torch::Tensor output_row_offsets = torch::zeros({(n + 1) * nonzeros.size(0)}, index_options);
    torch::Tensor output_column_indices = torch::zeros({column_indices_out_size}, index_options);

    std::vector<torch::Tensor> out_vector;
    out_vector.push_back(output_values);
    out_vector.push_back(output_row_offsets);
    out_vector.push_back(output_column_indices);

    for(int idx = 0; idx < replication; idx++) {
        int batch_index = idx / num_heads;
        int nonzero = nonzeros[batch_index].item<int>();
        // (Possibly) get a temporary buffer to work in.
        torch::Tensor workspace = allocate_transpose_workspace_many(&handle, m, n, nonzero, 
                                                            values, 
                                                            row_offsets, 
                                                            column_indices, 
                                                            output_values, 
                                                            output_row_offsets, 
                                                            output_column_indices);

        // Launch the kernel.
        CUSPARSE_CALL(hipsparseCsr2cscEx2(
            handle, m, n, nonzero, 
            values.data_ptr<float>() + batch_index * max_nonzeros, 
            row_offsets.data_ptr<int>() + batch_index * (n + 1),
            column_indices.data_ptr<int>(), 
            output_values.data_ptr<float>(), 
            output_row_offsets.data_ptr<int>(), 
            output_column_indices.data_ptr<int>(),
            HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
            HIPSPARSE_CSR2CSC_ALG1, workspace.data_ptr<float>()));
    }

    return out_vector;
}