#include <sputnik/sputnik.h>
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <c10/cuda/CUDAStream.h>

#define CUDA_CALL(code)                                     \
  do {                                                      \
    hipError_t status = code;                              \
    std::string err = hipGetErrorString(status);           \
    CHECK_EQ(status, hipSuccess) << "CUDA Error: " << err; \
  } while (0)

torch::Tensor spmm(int m, int k, int n, int nonzeros,
               torch::Tensor row_indices, 
               torch::Tensor values,
               torch::Tensor row_offsets, 
               torch::Tensor column_indices,
               torch::Tensor dense_matrix, 
               torch::Tensor bias) {
    at::cuda::CUDAStream torch_stream = at::cuda::getCurrentCUDAStream();
    hipStream_t stream = torch_stream.stream();

    auto options = torch::TensorOptions()
                                        .dtype(torch::kFloat32)
                                        .layout(torch::kStrided)
                                        .device(torch::kCUDA, 0)
                                        .requires_grad(true);
    torch::Tensor out = torch::zeros({m, n}, options);
    
    CUDA_CALL(sputnik::CudaSpmmBiasRelu(m, k, n, nonzeros, 
                                row_indices.data_ptr<int>(), 
                                values.data_ptr<float>(),
                                row_offsets.data_ptr<int>(), 
                                column_indices.data_ptr<int>(),
                                dense_matrix.data_ptr<float>(),
                                bias.data_ptr<float>(), 
                                out.data_ptr<float>(), 
                                stream));
    hipDeviceSynchronize();
    
    return out;
}
