#include <sputnik/sputnik.h>
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <c10/cuda/CUDAStream.h>

#define CUDA_CALL(code)                                     \
  do {                                                      \
    hipError_t status = code;                              \
    std::string err = hipGetErrorString(status);           \
    CHECK_EQ(status, hipSuccess) << "CUDA Error: " << err; \
  } while (0)

 torch::Tensor sddmm(int m, int k, int n, int nonzeros,
                           torch::Tensor row_indices,
                           torch::Tensor row_offsets,
                           torch::Tensor column_indices,
                           torch::Tensor lhs_matrix,
                           torch::Tensor rhs_matrix,
                           torch::Tensor output_values) {
    at::cuda::CUDAStream torch_stream = at::cuda::getCurrentCUDAStream();
    hipStream_t stream = torch_stream.stream();
    
    CUDA_CALL(sputnik::CudaSddmm(m, k, n, nonzeros, 
                                row_indices.data_ptr<int>(), 
                                row_offsets.data_ptr<int>(), 
                                column_indices.data_ptr<int>(),
                                lhs_matrix.data_ptr<float>(), 
                                rhs_matrix.data_ptr<float>(), 
                                output_values.data_ptr<float>(), 
                                stream));
    hipDeviceSynchronize();
    
    return output_values;
}
