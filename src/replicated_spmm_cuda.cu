#include <sputnik/sputnik.h>
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <c10/cuda/CUDAStream.h>

#define CUDA_CALL(code)                                     \
  do {                                                      \
    hipError_t status = code;                              \
    std::string err = hipGetErrorString(status);           \
    CHECK_EQ(status, hipSuccess) << "CUDA Error: " << err; \
  } while (0)

torch::Tensor replicated_spmm(int replication, int m, int k, int n, int nonzeros,
               torch::Tensor row_indices, 
               torch::Tensor values,
               torch::Tensor row_offsets, 
               torch::Tensor column_indices,
               torch::Tensor dense_matrix, 
               torch::Tensor bias) {
    at::cuda::CUDAStream torch_stream = at::cuda::getCurrentCUDAStream();
    hipStream_t stream = torch_stream.stream();

    auto options = torch::TensorOptions()
                                        .dtype(torch::kFloat32)
                                        .layout(torch::kStrided)
                                        .device(torch::kCUDA, 0)
                                        .requires_grad(true);
    torch::Tensor out = torch::zeros({replication, m, n}, options);

    for(int idx = 0; idx < replication; idx++) {
        CUDA_CALL(sputnik::CudaSpmmBiasRelu(m, k, n, nonzeros, 
                                row_indices.data_ptr<int>(), 
                                values.data_ptr<float>() + nonzeros * idx,
                                row_offsets.data_ptr<int>(), 
                                column_indices.data_ptr<int>(),
                                dense_matrix.data_ptr<float>() + k * n * idx,
                                bias.data_ptr<float>(), 
                                out.data_ptr<float>() + m * n * idx, 
                                stream));
    }
    hipDeviceSynchronize();
    
    return out;
}
