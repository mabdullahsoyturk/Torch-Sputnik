#include <sputnik/sputnik.h>
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <c10/cuda/CUDAStream.h>

#define CUDA_CALL(code)                                     \
  do {                                                      \
    hipError_t status = code;                              \
    std::string err = hipGetErrorString(status);           \
    CHECK_EQ(status, hipSuccess) << "CUDA Error: " << err; \
  } while (0)

torch::Tensor softmax(int m, int n, int nonzeros,
                      torch::Tensor values,
                      torch::Tensor row_indices,
                      torch::Tensor row_offsets,
                      torch::Tensor column_indices,
                      torch::Tensor output_values) {
    at::cuda::CUDAStream torch_stream = at::cuda::getCurrentCUDAStream();
    hipStream_t stream = torch_stream.stream();

    CUDA_CALL(sputnik::SparseSoftmax(m, n, nonzeros, 
                                values.data_ptr<float>(),
                                row_indices.data_ptr<int>(), 
                                row_offsets.data_ptr<int>(), 
                                column_indices.data_ptr<int>(),
                                output_values.data_ptr<float>(), 
                                stream));
    hipDeviceSynchronize();
    
    return output_values;
}
