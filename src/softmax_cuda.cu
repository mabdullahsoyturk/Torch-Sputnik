#include <sputnik/sputnik.h>
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <c10/cuda/CUDAStream.h>
#include "error_check.h"

torch::Tensor softmax(int m, int n, int nonzeros,
                      torch::Tensor values,
                      torch::Tensor row_indices,
                      torch::Tensor row_offsets,
                      torch::Tensor column_indices) {
    at::cuda::CUDAStream torch_stream = at::cuda::getCurrentCUDAStream();
    hipStream_t stream = torch_stream.stream();

    int dim_offset = values.dim() - 1;
    int replication = dim_offset == 1 ? values.size(0) : 1;

    auto options = torch::TensorOptions()
                                        .dtype(torch::kFloat32)
                                        .layout(torch::kStrided)
                                        .device(torch::kCUDA, 0)
                                        .requires_grad(true);
    torch::Tensor out = torch::zeros_like(values, options);

    for (int idx = 0; idx < replication; ++idx) {
      CUDA_CALL(sputnik::SparseSoftmax(m, n, nonzeros, 
                                values.data_ptr<float>() + nonzeros * idx,
                                row_indices.data_ptr<int>() + m * idx, 
                                row_offsets.data_ptr<int>() + (m + 1) * idx, 
                                column_indices.data_ptr<int>() + nonzeros * idx,
                                out.data_ptr<float>() + nonzeros * idx, 
                                stream));      
    }

    hipDeviceSynchronize();
    
    return out;
}
